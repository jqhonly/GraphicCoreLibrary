#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#include "SpatialTransform.cuh"

extern "C"
int Resize_Host(unsigned char*h_src, int srcWidth, int srcHeight, unsigned char *h_dst, int dstWidth, int dstHeight, int deviceid)
{
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(deviceid);
	if (cudaStatus != hipSuccess)
		return cudaStatus;

	unsigned char * d_src;
	unsigned char * d_dst;
	hipMalloc((void**)&d_src, srcWidth * srcHeight * 4 * sizeof(unsigned char));
	hipMalloc((void**)&d_dst, dstHeight * dstWidth * 4 * sizeof(unsigned char));

	hipMemcpy(d_src, h_src, srcWidth * srcHeight * 4 * sizeof(unsigned char), hipMemcpyHostToDevice);
	int uint = 16;//Don't change!
	dim3 grid((dstWidth + uint - 1) / uint, (dstHeight + uint - 1) / uint);
	dim3 block(uint, uint);

	Resize_Device << <grid, block >> >(d_src, srcWidth, srcHeight, d_dst, dstWidth, dstHeight);

	hipMemcpy(h_dst, d_dst, dstHeight * dstWidth * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(d_dst);
	hipFree(d_src);

	cudaStatus = hipGetLastError();
	return cudaStatus;
}