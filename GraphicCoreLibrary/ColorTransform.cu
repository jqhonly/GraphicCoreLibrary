#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
//#include <hip/hip_vector_types.h>

#include "ColorTransform.cuh"

extern "C"
int YV12toARGB32(unsigned char* d_YV12, unsigned char* d_RGBA32, int width, int height, int deviceid)
{
	hipError_t cudaStatus;
	
	cudaStatus = hipSetDevice(deviceid);
	if (cudaStatus != hipSuccess)
		return cudaStatus;
	dim3 block(32, 8);
	int gridx = (width + 2 * block.x - 1) / (2 * block.x);
	int gridy = (height + 2 * block.y - 1) / (2 * block.y);
	dim3 grid(gridx, gridy);
	
	YV12ToARGB_FourPixel << <grid, block >> >(d_YV12, (unsigned int*)d_RGBA32, width, height);
	
	cudaStatus = hipGetLastError();
	
	return cudaStatus;
}